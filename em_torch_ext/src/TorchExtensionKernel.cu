#include "hip/hip_runtime.h"
/**********************************************************************************************************************
 * Name: TorchExtensionKernel.cpp
 * Author: Matin Raayai Ardakani
 * Email: matinraayai@seas.harvard.edu
 * Where the CUDA magic happens for the em_pre_cuda Python package.
 * Based on the code from Pytorch's tutorials: https://github.com/pytorch/extension-cpp
 **********************************************************************************************************************/
#include "TorchExtensionKernel.h"

/*
 * A helper function used in each CUDA thread that returns the idx if if idx > minIdx and idx < maxIdx. If not,
 * it will "reflect" the returned index so that it falls between the minimum and maximum range.
 * This helps with applying a 3D median filter while "reflecing" the boundries.
 * @param idx the index
 * @param minIdx the lower bound of the 1D tensor
 * @param maxIdx the upper bound of the 1D tensor
 * @return the index of the element, safe to access the intended 1D tensor.
 */
__device__ __forceinline__ int clamp_mirror(int idx, int minIdx, int maxIdx)
{
    if(idx < minIdx) return clamp_mirror(minIdx + (minIdx - idx), minIdx, maxIdx);
    else if(idx > maxIdx) return clamp_mirror(maxIdx - (idx - maxIdx), minIdx, maxIdx);
    else return idx;
}

template<typename scalar_t>
__device__ __host__ scalar_t get_median_of_array(scalar_t* vector, int vSize)
{
    for (int i = 0; i < vSize; i++) {
    for (int j = i + 1; j < vSize; j++) {
        if (vector[i] > vector[j]) {
            scalar_t tmp = vector[i];
            vector[i] = vector[j];
            vector[j] = tmp;
        }
    }}
    return vector[vSize / 2];
}

template<typename scalar_t>
__global__
void __median_3d(scalar_t* __restrict__ input, scalar_t* __restrict__ output, int dimX, int dimY,
                 int dimZ, int radX, int radY, int radZ)
{
    auto get_1d_idx = [&] (int32_t x, int32_t y, int32_t z) {
        return clamp_mirror(z, 0, dimZ - 1) * dimY * dimX +
               clamp_mirror(y, 0, dimY - 1) * dimX + clamp_mirror(x, 0, dimX - 1);
    };

    const int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int col_idx = blockIdx.y * blockDim.y + threadIdx.y;
    const int sht_idx = blockIdx.z * blockDim.z + threadIdx.z;

    scalar_t windowVec[MAX_GPU_ARRAY_LEN] = {0.};
    int vSize = 0;

    if (col_idx < dimX && row_idx < dimY && sht_idx < dimZ) {
        for (int z = -radZ; z <= radZ; z++)
        for (int y = -radY; y <= radY; y++)
        for (int x = -radX; x <= radX; x++)
            windowVec[vSize++] = input[get_1d_idx(x + row_idx, y + col_idx, z + sht_idx)];
        output[get_1d_idx(row_idx, col_idx, sht_idx)] = get_median_of_array(windowVec, vSize);
    }
}

template<typename scalar_t>
__device__ __forceinline__
scalar_t __patch_distance (const int A_x, const int A_y, const int B_x, const int B_y,
                           const int im_row, const int im_col, const int im_chan,
                           const int patch_sz, scalar_t *img1, scalar_t *img2, int metric){
    scalar_t dist = 0, temp_h;
    int c, x, y, count = 0;
    /* only move around patchB */
    int pre = im_col * im_chan;
    scalar_t patch_sum = 0;

    switch(metric) {
        case 0: // L1
            for(y = -patch_sz; y <= patch_sz; y++) {
            for(x = -patch_sz; x <= patch_sz; x++) {
                if((A_x + x) >= 0 && (A_y + y) >= 0 && (A_x + x) < im_row && (A_y + y) < im_col &&
                   (B_x + x) >= 0 && (B_y + y) >= 0 && (B_x + x) < im_row && (B_y + y) < im_col) {
                    for(c = 0; c < im_chan; c++) {
                        temp_h = img1[(A_x + x)*pre + (A_y + y)*im_chan + c] -
                                 img2[(B_x + x)*pre + (B_y + y)*im_chan + c];
                        dist += fabsf(temp_h);
                        count++;
                    }
                }
            }}
            break;
        case 1: // relative L1
            for(y=-patch_sz; y<=patch_sz; y++){
                for(x=-patch_sz; x<=patch_sz; x++){
                    if((A_x + x)>=0 && (A_y + y)>=0 && (A_x + x)<im_row && (A_y + y)<im_col
                       && (B_x + x)>=0 && (B_y + y)>=0 && (B_x + x)<im_row && (B_y + y)<im_col){
                        for(c=0; c<im_chan; c++){
                            temp_h = img1[(A_x + x)*pre + (A_y + y)*im_chan + c] -
                                     img2[(A_x + x)*pre + (A_y + y)*im_chan + c];
                            dist += fabsf(temp_h);
                            patch_sum += img1[(A_x + x)*pre + (A_y + y)*im_chan + c];
                            //dist+=temp_h*temp_h;
                            count++;
                        }
                    }
                }
            }
            dist = dist/patch_sum;
            break;
    }
    return dist/count;
}
////

template<typename scalar_t>
__global__
void __idm_dist(scalar_t* img1, scalar_t* img2, scalar_t* dis,
                int dimX, int dimY, int im_chan,
                int outX, int outY,
                int patch_sz, int warp_sz, int patch_step, int metric) {
    /* assume same size img */
    scalar_t best_dis, temp;
    int xx, yy;
    const int dis_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int dis_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = patch_step * dis_x;
    const int y = patch_step * dis_y;
    /* 3) Return distance */
    best_dis = std::numeric_limits<scalar_t>::max();
    if (dis_x < outX && dis_y < outY) {
        for (xx = x - warp_sz; xx <= x + warp_sz; xx++) {
            for (yy = y - warp_sz; yy <= y + warp_sz; yy++) {
                if (xx >= 0 && yy >= 0 && xx < dimX && yy < dimY) {
                    temp = __patch_distance(x, y, xx, yy, dimX, dimY, im_chan,
                                            patch_sz, img1, img2, metric);
                    if (temp < best_dis)
                        best_dis = temp;
                }
            }
        }
        dis[dis_x * outY + dis_y] = best_dis;
    }
}



at::Tensor cuda_median_3d(const at::Tensor& tensor, const int radX, const int radY, const int radZ) {
    at::Tensor out = at::zeros_like(tensor);
    const int dimX = tensor.size(2), dimY = tensor.size(1), dimZ = tensor.size(0);
    const dim3 blockDim(BLOCK_DIM_LEN, BLOCK_DIM_LEN, BLOCK_DIM_LEN);
    const dim3 gridDim(
            (dimX / blockDim.x + ((dimX % blockDim.x) ? 1 : 0)),
            (dimY / blockDim.y + ((dimY % blockDim.y) ? 1 : 0)),
            (dimZ / blockDim.z + ((dimZ % blockDim.z) ? 1 : 0)));

    AT_DISPATCH_FLOATING_TYPES(tensor.scalar_type(), "__median_3d", ([&] {
        __median_3d<scalar_t><<<gridDim, blockDim>>>(
                tensor.data_ptr<scalar_t>(),
                out.data_ptr<scalar_t>(),
                dimX,
                dimY,
                dimZ,
                radX,
                radY,
                radZ);
    }));
    return out;
}

at::Tensor cuda_idm(const at::Tensor& tensor1,
                    const at::Tensor& tensor2,
                    int patch_size,
                    int warp_size,
                    int patch_step,
                    int metric) {
    const int numChannel = tensor1.size(2), dimY = tensor1.size(1), dimX = tensor1.size(0);
    const int outY = static_cast<int>(ceil(dimY) / static_cast<float>(patch_step));
    const int outX = static_cast<int>(ceil(dimX) / static_cast<float>(patch_step));
    at::Tensor out = at::zeros({outX, outY},
                               at::TensorOptions().dtype(at::kFloat).device(at::kCUDA));
    const dim3 blockDim(16, 16);
    const dim3 gridDim((outX / blockDim.x + ((outX % blockDim.x) ? 1 : 0)),
                       (outY / blockDim.y + ((outY % blockDim.y) ? 1 : 0)));

    AT_DISPATCH_FLOATING_TYPES(tensor1.scalar_type(), "__idm_dist", ([&] {
        __idm_dist<scalar_t><<<gridDim, blockDim>>>(
                tensor1.data_ptr<scalar_t>(),
                tensor2.data_ptr<scalar_t>(),
                out.data_ptr<scalar_t>(),
                dimX,
                dimY,
                numChannel,
                outX,
                outY,
                patch_size,
                warp_size,
                patch_step,
                metric);
    }));
    return out;
}